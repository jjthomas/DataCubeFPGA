#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

#include <hiprand/hiprand.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__global__ void run(uint8_t *input, uint8_t group_size, uint32_t num_input_lines, uint32_t *output) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t first_group_idx = MIN(index / group_size, group_size - 1);
  uint32_t second_group_idx = index % group_size;
  uint32_t *our_output = output + index * 512;
  uint8_t *input_ptr = input;

  uint32_t counts[512] = {0};

  for (uint32_t i = 0; i < num_input_lines; i++) {
    uint32_t metric = 0;
    for (uint32_t j = 0; j < 4; j++) {
      metric = metric | (input_ptr[j] << (j * 8));
    }
    input_ptr += 4;
    uint8_t counts_idx = input_ptr[first_group_idx] | (input_ptr[group_size + second_group_idx] << 4);
    counts[2 * counts_idx] += metric;
    counts[2 * counts_idx + 1]++;
    input_ptr += 2 * group_size;
  }
  for (uint32_t i = 0; i < 512; i++) {
    our_output[i] = counts[i];
  }
}

int main(int argc, char **argv) {
  assert(hipSetDevice(0) == hipSuccess);

  uint32_t group_size = sqrt(NUM_THREADS);
  uint32_t num_lines = 1000000;
  uint32_t input_size = (sizeof(uint32_t) + 2 * sizeof(uint8_t) * group_size) * num_lines;
  printf("group size: %d, input size: %d\n", group_size, input_size);
  uint8_t *input_dev;
  uint32_t *output_dev;
  assert(hipMalloc((void **) &output_dev, 512 * sizeof(uint32_t) * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, input_size) == hipSuccess);
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
  hiprandGenerate(prng, (uint32_t *)input_dev, input_size / 4);
  // cudaMemset(input_dev, 0, input_size);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>(input_dev, group_size, num_lines, output_dev);
  assert(hipDeviceSynchronize() == hipSuccess);
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;

  uint32_t *output = new uint32_t[512 * NUM_THREADS];
  hipMemcpy(output, output_dev, 512 * sizeof(uint32_t) * NUM_THREADS, hipMemcpyDeviceToHost);
  double group_correction = pow((double)group_size / 40, 2); // group size on FPGA is only ~40
  printf("%.2f MB/s (%.2f MB/s), random byte: %d\n",
    input_size / 1000000.0 / secs,
    input_size / 1000000.0 / secs * group_correction,
    output[1]); // output[rand() % (512 * NUM_THREADS)]
  return 0;
}
