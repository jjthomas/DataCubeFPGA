#include "hip/hip_runtime.h"
#include <stdint.h>
#include <sys/time.h>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>

using namespace std;

#define NUM_SMS 110
// must be power of two
#define BLOCK_SIZE 256
// must be power of two
#define NUM_THREADS_PER_SM 2048
#define NUM_BLOCKS_PER_SM (NUM_THREADS_PER_SM / BLOCK_SIZE)
#define NUM_BLOCKS (NUM_SMS * NUM_BLOCKS_PER_SM)
#define NUM_THREADS (NUM_THREADS_PER_SM * NUM_SMS)

#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__global__ void run(uint8_t *input, uint8_t group_size, uint32_t num_input_lines, uint32_t *output) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t first_group_idx = MIN(index / group_size, group_size - 1);
  uint32_t second_group_idx = index % group_size;
  uint32_t input_line_size = 4 + 2 * group_size; // 32 bits for metric and two groups
  uint32_t output_size = 256 * 2; // word size of 4 means 256 slots, each with 32 bits for
  // metric and 32 bits for count
  uint32_t *our_output = output + index * output_size;
  uint8_t *input_ptr = input;

  uint32_t counts[output_size] = {0};

  for (uint32_t i = 0; i < num_input_lines; i++) {
    uint32_t metric = 0;
    for (uint32_t j = 0; j < 4; j++) {
      metric = metric | (input_ptr[j] << (j * 8));
    }
    input_ptr += 4;
    uint8_t counts_idx = input_ptr[first_group_idx] | (input_ptr[group_size + second_group_idx] << 4);
    counts[2 * counts_idx] += metric;
    counts[2 * counts_idx + 1]++;
    input_ptr += 2 * group_size;
  }
  for (uint32_t i = 0; i < output_size; i++) {
    our_output[i] = counts[i];
  }
}

int main(int argc, char **argv) {
  hipSetDevice(0);

  uint32_t group_size = sqrt(NUM_THREADS);
  uint32_t num_lines = 10000000;
  uint32_t input_size = (sizeof(uint32_t) + 2 * sizeof(uint8_t) * group_size) * num_lines;
  uint8_t *input_dev;
  uint32_t *output_dev;
  assert(hipMalloc((void **) &output_dev, 256 * 2 * sizeof(uint32_t) * NUM_THREADS) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, input_size) == hipSuccess);

  struct timeval start, end, diff;
  gettimeofday(&start, 0);
  run<<<NUM_BLOCKS, BLOCK_SIZE>>>(input_dev, group_size, num_lines, output_dev);
  hipDeviceSynchronize();
  gettimeofday(&end, 0);
  timersub(&end, &start, &diff);
  double secs = diff.tv_sec + diff.tv_usec / 1000000.0;
  printf("%.2f MB/s\n", input_size / 1000000.0 / secs);
  return 0;
}
